#include "ksscan.cuh"
#include "real.h"
#include "gpuerrchk.cuh"
#include "assert.h"
#include <iostream>
void test(){
	real X[]={3, 1, 7, 0 ,4 ,1 ,6, 3};
	real Y[8];
	real* d_X;
	real* d_Y;

	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*8));
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*8,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*8 ));
	ksscan(d_X,d_Y,8);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*8,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));
	assert(Y[3]==11);
	assert(Y[7]==25);
}

int main(){
	test();
	std::cout << "success!!!"<<std::endl;
}
