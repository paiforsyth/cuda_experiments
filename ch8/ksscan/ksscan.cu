#include "hip/hip_runtime.h"
#include "real.h"
#include "math.h"

#define SECTION_SIZE 512
//ACTUALLY THIS SEEMS WRONG: WE DO NOT KNOW THE ORDER OF OPERATIONS OF THE ADDING.  NEED TO DOUBLE BUUFER THE ARRAY XY TO GUARATEE THAT THIS WORKS.
__global__ void ksscan_kernel(real* X, real* Y, int inputsize){
	__shared__ real XY[SECTION_SIZE];
	int i =blockIdx.x*blockDim.x+threadIdx.x;
	if (i < inputsize){
		XY[threadIdx.x]=X[i]; 
		for (int stride=1; stride<blockDim.x; stride*=2 ){
			__syncthreads();
			if (threadIdx.x >= stride) XY[threadIdx.x]+= XY[threadIdx.x-stride];
		}
		Y[i]=XY[threadIdx.x];
	}
}

void ksscan(real* d_X, real* d_Y,int inputsize){
	ksscan_kernel<<<ceil(inputsize/ (real) SECTION_SIZE),SECTION_SIZE>>>(d_X,d_Y,inputsize);
}



