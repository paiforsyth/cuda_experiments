#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <assert.h>
#include <memory>
#include <random>
#include "gMat.cuh"
#include "gpuerrchk.cuh"
#include "real.h"
__global__ void matMulKernel(unsigned int num_rows, real* data, unsigned int* col_index, unsigned int row_p, real* x, real* y){
	unsigned int row =blockIdx.x * blockDim.x +threadIdx.x;
	if (row < num_rows){
		real dot=0;
		unsigned int row_start= row_p[row];
		unsigned int row_end= row_p[row+1];
		for (unsigned int elem=row_start; elem<row_end; ++elem){
			dot+=data[elem]*x[col_index[elem]];
		}	
		y[row]+=dot;
	}
}


gMat::gMat(std::vector<real> datavector, int r, int c,int devnumber): rows{r}, cols{c} {
	assert(datavector.size() == rows*cols );
	int size=sizeof(real)*rows*cols;

	hipDeviceProp_t dev_prop;	
	hipGetDeviceProperties(&dev_prop,devnumber);
	assert(dev_prop.totalGlobalMem>size);

	gpuErrchk(hipMalloc( (void**)&d_data, size));
	gpuErrchk(hipMemcpy(d_data, datavector.data(), size, hipMemcpyHostToDevice));
}

void gMat::cleanup(){
	gpuErrchk(hipFree(d_data));
}



int gMat::getrows(){ return rows;}
int gMat::getcols(){ return cols;}

real gMat::entry(int i, int j){
	assert(i>=0 && i<rows);
	assert(j>=0 && j<cols);
	real val=0;

	hipMemcpy(&val, &d_data[i*cols+j],sizeof(real),hipMemcpyDeviceToHost );
	return val;
}


std::vector<real> gMat::tovector(){
	real* h_data=new real[rows*cols];
	int size=rows*cols*sizeof(real);
	hipMemcpy(h_data,d_data,size,hipMemcpyDeviceToHost);
	std::vector<real> v;
	for (int i=0; i<rows; ++i) for (int j=0; j<cols; ++j){
		v.push_back(h_data[i*cols+j]);
	}
	delete[] h_data;
	return v;
}	

gMat randgMat(int r, int c){
	std::vector<real> datavector;
	std::default_random_engine re;
	std::uniform_real_distribution<real> dist{0.0,1.0};
	for (int i=0; i<r*c; i++) datavector.push_back(dist(re));
	return gMat(datavector,r,c);
}

gMat eye(int r){
	std::vector<real> datavector;
	for (int i=0; i!=r; ++i) for(int j=0; j!=r; ++j) 
		if(i==j) 
			datavector.push_back(1.0);
		else
			datavector.push_back(0.0);
	return gMat(datavector,r,r);
}

void prod(const gMat& A, const gMat& B, gMat& P) {
	int w=16;
	int mem=2*w*w*sizeof(real);
	prod(A,B,P,mem);
}

void prod(const gMat& A, const gMat& B, gMat& P, int mem){
	assert(A.cols==B.rows && A.rows==P.rows && B.cols==P.cols);
	int tile_size= floor(sqrt( mem/2/sizeof(real)));
	assert(tile_size>0);	

	matMulKernel<<<dim3(ceil(B.cols/ (real) tile_size), ceil( A.rows/ (real) tile_size)), dim3(tile_size,tile_size), mem>>>(A.d_data, B.d_data, P.d_data,A.rows, A.cols, B.cols, tile_size );
	gpuErrchk( hipPeekAtLastError() );	
}

void prodv2(const gMat& A, const gMat& B, gMat& P, int tile_size, int stretch){
	assert(A.cols==B.rows && A.rows==P.rows && B.cols==P.cols);
	assert(tile_size>0 && stretch>0);
	int mem= 2*tile_size*tile_size*stretch*sizeof(real);
	matMulKernelv2 <<<dim3(ceil(B.cols/ (real) tile_size), ceil( A.rows/ (real) tile_size)), dim3(tile_size,tile_size), mem>>>(A.d_data, B.d_data, P.d_data, A.rows, A.cols, B.cols, tile_size, stretch);
	gpuErrchk( hipPeekAtLastError() );	
}

std::ostream& operator<<(std::ostream& os,  gMat& gm)  
{  
	std::vector<real> v=gm.tovector();
	for (int i=0; i<gm.rows; ++i){
		for (int j=0; j<gm.cols; ++j){
			os << v[i*gm.cols+j] << ",";
		}
		os << "\n";
	}
	return os;  

}  


