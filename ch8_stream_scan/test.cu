#include "gpuerrchk.cuh"
#include "assert.h"
#include "real.h"
#include "ch8_stream_scan.cuh"
#include <iostream>
void test(){
	constexpr size_t DAT_SIZE=2048;
	real X[DAT_SIZE];
	for (int i=0; i< DAT_SIZE; ++i) X[i]=1;
	real Y[DAT_SIZE];
	real* d_X;
	real* d_Y;
	real* d_S;
	int* d_flags;
	int* d_DCounter;
	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*DAT_SIZE ));	
	gpuErrchk(hipMalloc((void**) &d_S,sizeof(real)*DAT_SIZE ));
	gpuErrchk(hipMalloc((void**) &d_flags, sizeof(int)*DAT_SIZE ));
	gpuErrchk(hipMalloc((void**) &d_DCounter,sizeof(int)));
	gpuErrchk( hipMemset(d_flags,0, sizeof(int)*DAT_SIZE) );
	gpuErrchk( hipMemset(d_DCounter,0, sizeof(int)) );	
	ch8_stream_scan(d_X,d_Y,d_S,d_flags,d_DCounter,DAT_SIZE, DAT_SIZE/1024, 1024);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*DAT_SIZE,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));
	gpuErrchk(hipFree(d_S));
	gpuErrchk(hipFree(d_flags));
	gpuErrchk(hipFree(d_DCounter));
	
	for (int j=0; j<DAT_SIZE; ++j){
		//std::cout<< "j=" << j << " Y[j]=" << Y[j] <<std::endl;
		assert(Y[j]==j+1);
	}
}


void test2(){
	constexpr size_t DAT_SIZE=2048*100;
	real X[DAT_SIZE];
	for (int i=0; i< DAT_SIZE; ++i) X[i]=1;
	real Y[DAT_SIZE];
	real* d_X;
	real* d_Y;
	real* d_S;
	int* d_flags;
	int* d_DCounter;
	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*DAT_SIZE ));	
	gpuErrchk(hipMalloc((void**) &d_S,sizeof(real)*DAT_SIZE ));
	gpuErrchk(hipMalloc((void**) &d_flags, sizeof(int)*DAT_SIZE ));
	gpuErrchk(hipMalloc((void**) &d_DCounter,sizeof(int)));
	gpuErrchk( hipMemset(d_flags,0, sizeof(int)*DAT_SIZE) );
	gpuErrchk( hipMemset(d_DCounter,0, sizeof(int)) );	
	ch8_stream_scan(d_X,d_Y,d_S,d_flags,d_DCounter,DAT_SIZE, DAT_SIZE/1024, 1024);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*DAT_SIZE,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));
	gpuErrchk(hipFree(d_S));
	gpuErrchk(hipFree(d_flags));
	gpuErrchk(hipFree(d_DCounter));
	
	for (int j=0; j<DAT_SIZE; ++j){
		//std::cout<< "j=" << j << " Y[j]=" << Y[j] <<std::endl;
		assert(Y[j]==j+1);
	}
}

void test3(){
	std::cout<<"in test 3 body" <<std::endl<<std::flush;
	constexpr size_t DAT_SIZE=1024*1024;
std::cout<<"Before X allocation" <<std::endl;
	real* X=new real[DAT_SIZE];
	std::cout<<"after X allocation" <<std::endl;
	for (size_t i=0; i< DAT_SIZE; ++i) X[i]=1;
	std::cout<<"after X initialization" <<std::endl;
	real Y[DAT_SIZE];
	real* d_X;
	real* d_Y;
	real* d_S;
	int* d_flags;
	int* d_DCounter;
	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*DAT_SIZE));
	std::cout << "before d_X allocated" << std::endl;
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*DAT_SIZE ));	
	gpuErrchk(hipMalloc((void**) &d_S,sizeof(real)*DAT_SIZE/1024 ));
	gpuErrchk(hipMalloc((void**) &d_flags, sizeof(int)*DAT_SIZE/1024 ));
	gpuErrchk(hipMalloc((void**) &d_DCounter,sizeof(int)));
	gpuErrchk( hipMemset(d_flags,0, sizeof(int)*DAT_SIZE/1024) );
	gpuErrchk( hipMemset(d_DCounter,0, sizeof(int)) );	
	std::cout<< "Before scan called" <<std::endl;
	ch8_stream_scan(d_X,d_Y,d_S,d_flags,d_DCounter,DAT_SIZE, DAT_SIZE/1024, 1024);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*DAT_SIZE,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));
	gpuErrchk(hipFree(d_S));
	gpuErrchk(hipFree(d_flags));
	gpuErrchk(hipFree(d_DCounter));
	delete[] X;
	for (int j=0; j<DAT_SIZE; ++j){
		//std::cout<< "j=" << j << " Y[j]=" << Y[j] <<std::endl;
		assert(Y[j]==j+1);
	}
}



int main(){
	test();
	
	std::cout<< "Success in test 1!!!" <<std::endl;
	test2();
	
	std::cout<< "Success in test 2!!!" <<std::endl;
	test3();
	std::cout<< "Success in test 3!!!" <<std::endl;
}

