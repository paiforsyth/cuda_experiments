#include "hip/hip_runtime.h"
#include <math.h>
#include "gpuerrchk.cuh"
#include "real.h"
#include "next_pow_2.h"
#include <assert.h>
#include <iostream>
__global__ void ch8_stream_scan_kernel1(real* X, real* Y,real* S,int* flags,int* DCounter, unsigned int S_length, unsigned int treesize){
	extern __shared__ real XY[];
	
	__shared__ int sbid;
	if (threadIdx.x == 0)
		sbid=atomicAdd(DCounter, 1);
	__syncthreads();
	const int bid=sbid;

	int i=2*bid*blockDim.x+threadIdx.x;
	 XY[threadIdx.x]=X[i];
	 XY[threadIdx.x+blockDim.x]=X[i+blockDim.x];	
	
	//up-sweep
	for (int stride=1; stride <= blockDim.x; stride*=2){
		__syncthreads();
		int index= 2*(threadIdx.x+1)*stride -1;
		if (index< treesize) XY[index]+=XY[index-stride];
	}


	//down-sweep
	for (int stride=treesize/2; stride>=1; stride/=2){
		__syncthreads();
		int index = 2*(threadIdx.x+1)*stride-1;
		if (index+stride < treesize){
			XY[index+stride]+=XY[index];
		}
	}

	//domino
	__syncthreads(); 
	__shared__ float previous_sum;
	 if (threadIdx.x == blockDim.x-1  ){
		if (bid > 0){
			while(atomicAdd(&flags[bid-1],0 ) == 0 ) {;}
			previous_sum=S[bid-1];
			S[bid]=previous_sum+XY[2*blockDim.x-1];
			__threadfence();
			atomicAdd(&flags[bid],1);
		}
		else{
			previous_sum=0;
			S[0]=XY[2*blockDim.x-1];
			__threadfence();
			atomicAdd(&flags[0],1);
		}
	 }

	 __syncthreads();
	 XY[threadIdx.x]+=previous_sum;
	 XY[threadIdx.x+blockDim.x]+=previous_sum;
	 Y[i]= XY[threadIdx.x];
	 Y[i+blockDim.x]= XY[threadIdx.x+blockDim.x]; 
}




//treesize is assumed to be a power of 2.  d_X and d_Y are assumed to be of length length. 
//also assume treesize*(S_length)=length
//also assume d_S is small enough to be scanned by one thread block.
//also assume d_S points to an array with length equal to S_length rounded to the next power of 2
void ch8_stream_scan(real* d_X, real* d_Y,real* d_S,int* flags, int* DCounter, size_t length, unsigned int S_length, unsigned int treesize){
	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop,0); //assume we are using device 0
	size_t share_mem=dev_prop.sharedMemPerBlock;
	int thread_limit= dev_prop.maxThreadsPerBlock;
	size_t max_per_block=share_mem/sizeof(real);
	assert(treesize<=max_per_block && treesize<=2*thread_limit);
	assert(treesize*(S_length)==length);
	ch8_stream_scan_kernel1<<<S_length, treesize/2, treesize*sizeof(real)>>>(d_X, d_Y, d_S, flags, DCounter, S_length, treesize);
	gpuErrchk(hipPeekAtLastError());
}
