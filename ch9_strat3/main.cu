#include "gpuerrchk.cuh"
#include "assert.h"
#include "real.h"
#include "ch9_strat3.cuh"
#include <iostream>
#include "rstring.h"
#include <string>
#define DAT_SIZE 10000000


void test(){
	std::string buffer_string = rstring(DAT_SIZE);
    const char* buffer= buffer_string.c_str();	
	unsigned int histo[]= {0,0,0,0,0,0,0};
	 char* d_buffer;
	unsigned int* d_histo;
	gpuErrchk(hipMalloc((void**) &d_buffer,sizeof(char)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_buffer,buffer,sizeof(char)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_histo,sizeof(int)*7 ));
	
	ch9_strat3(d_buffer,d_histo, (size_t) DAT_SIZE, 7);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(histo,d_histo,sizeof(int)*7,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_buffer));
	gpuErrchk(hipFree(d_histo));
}

int main(){
	test();
	std::cout<<"DONE!" <<std::endl;
}

