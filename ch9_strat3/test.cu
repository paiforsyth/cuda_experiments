#include "gpuerrchk.cuh"
#include "assert.h"
#include "real.h"
#include "ch9_strat3.cuh"
#include <iostream>
#define DAT_SIZE 41 
//void ch9_strat3(char* buffer, int* histo,size_t inputsize);
void test(){
	char buffer[]="programming massively parallel processors";
	unsigned int histo[]= {0,0,0,0,0,0,0};
	char* d_buffer;
	unsigned int* d_histo;
	gpuErrchk(hipMalloc((void**) &d_buffer,sizeof(char)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_buffer,buffer,sizeof(char)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_histo,sizeof(int)*7 ));
	
	ch9_strat3(d_buffer,d_histo, (size_t) DAT_SIZE,7);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(histo,d_histo,sizeof(int)*7,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_buffer));
	gpuErrchk(hipFree(d_histo));
	for (int i=0; i<7 ;i++){
		std::cout << "histogram bucket " << i << " has value " << histo[i] << std::endl;
	
	}

	

}

int main(){
	test();
}

