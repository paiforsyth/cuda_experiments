#include "hip/hip_runtime.h"
#include "gpuerrchk.cuh"
#include "real.h"

__global__ void ch9_strat3_kernel(char* buffer,unsigned int* histo, size_t inputsize,unsigned int num_bins){

	unsigned int tid = threadIdx.x +blockIdx.x* blockDim.x;
	extern __shared__ unsigned int histo_s[];
	for (unsigned int  j=threadIdx.x; j<num_bins; j+=blockDim.x)	
		histo_s[j]=0;		
	__syncthreads();
	for (unsigned int i=tid; i< inputsize; i+= blockDim.x*gridDim.x  ){
		int alphabet_position=buffer[i]-'a';
		if (alphabet_position >=0 && alphabet_position < 26)
			atomicAdd(&histo_s[alphabet_position/4], 1);
	}
	__syncthreads();
	for (unsigned int j=threadIdx.x; j<num_bins; j+=blockDim.x)
		atomicAdd(&histo[j], histo_s[j]);
}
void ch9_strat3(char* buffer, unsigned int* histo,size_t inputsize, unsigned int num_bins){
	//key question: how many thread blocks do we use??
	ch9_strat3_kernel<<<50, 100, num_bins * sizeof(int) >>>(buffer,histo,inputsize, num_bins); 
	gpuErrchk(hipPeekAtLastError());
}
