#include "hip/hip_runtime.h"
#include <iostream>
#include "gMat.cuh"
#include <vector>
#include "mytime.h"
#include <functional>
#include <math.h>
#include <stdlib.h> 
#include <string>
void manyMult(gMat& A, gMat& B, gMat& C,int iter,int mem){
	for (int i=0; i!=iter; ++i){
		prod(A,B,C,mem);
		prod(A,C,B,mem);
	}
}

void multstat(int mem, int row, int col){
	std::cout << "multiplying "  << row << " by "<< col <<" matrices.\n";
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	std::cout << "using "<< mem << " of "<<prop.sharedMemPerBlock << "shared memory\n";
	int w=floor(sqrt(mem/2/sizeof(float)));
	std::cout << "Tile sizes will be: "<< w<< " by "<< w << "\n";
	std::cout << "This amounts to: " << w*w << " threads \n";
	std::cout <<  ceil(row/ (float) w)*ceil(col/ (float) w) << " tiles will be used "<<std::endl;
}

int main(int argc, char *argv[]){
	int w;
	if ( argc >= 2) w=atoi(argv[1]);
	else w=20;
	int r=1000;
	int c=1000;
	gMat A=eye(r);
	gMat B=randgMat(r,c);
	gMat C=randgMat(r,c);
	A.name="A";
	B.name="B";
	C.name="C";
	int mem=2*w*w*sizeof(float);
	auto mm=std::bind(manyMult,A,B,C,300,mem);
	multstat(mem,r,c);
	mm();
	A.cleanup();
	B.cleanup();
	C.cleanup();
	 
}




void test1(){
	std::vector<float> adat={1,2,3,4};
	std::vector<float> bdat={2,0,0,2,0,2};
	std::vector<float> cdat={0,0,0,0,0,0};
	gMat A{adat,2,2};
	gMat B{bdat,2,3};
	gMat C{cdat,2,3};
	std::cout <<"A:\n" << A << std::flush;
	std::cout <<"B:\n" << B << std::flush;
	int mem=8*sizeof(float);
	prod(A,B,C,mem);
	std::cout <<"C:\n" << C << std::flush;
}
