#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <assert.h>
#include <memory>
#include <random>
#include "gMat.cuh"
#include "gpuerrchk.cuh"
__global__ void matMulKernel(float* A, float* B, float* P, int m, int  n, int s, int tile_size){
	//each thread in the block will be responsible for a different element of these arrays
	extern __shared__ float tilemem[];
	float* Atile=tilemem;
	float* Btile=&tilemem[tile_size*tile_size];

	int b1=blockIdx.x;
	int b2=blockIdx.y;
	int t1=threadIdx.x;
	int t2=threadIdx.y;

	int row=b1*tile_size+t1;// the indices of the element of P this thread is responsible for computing
	int col=b2*tile_size+t2;

	float Pvalue=0;
	//loop over the tiles necessary to compute the element of P of interest
	for (int j=0; j<ceil( n / (float) tile_size);j++){
		//collaborate with other threads to store the current tiles of A and B
		//this thread is responsible for storing  the (t_1,t_2) element of the (b_1,j) tile of A and the (j,b_2) tile of B
		if( (b1*tile_size+t1) < m && (j*tile_size+t2)<n )
			Atile[tile_size*t1 + t2]=A[ n*(b1*tile_size+t1) + j*tile_size  + t2 ]; // We access A[b1*tile_size+t1][j*tile_size+t2] using linearized indices (A has n columns)
		if( (j*tile_size+t1)<n && (b2*tile_size+t2)<s  )
			Btile[tile_size*t1 + t2]=B[ s*(j*tile_size+t1)  + b2*tile_size + t2 ]; //we access the B[j*tile_size+t1][b2*tile_size+t2] element using linearized indices (B has s columns)
		__syncthreads();

		//sum the elements of the current A and B tiles used in the computation of the element of P for which this thread is responsible
		for (int k=0; k<tile_size; k++)
			Pvalue += Atile[tile_size*t1 + k]*Btile[tile_size*k + t2];

		__syncthreads();

	}
	if(row<m && col<s) P[row*s + col]=Pvalue;

}

gMat::gMat(std::vector<float> datavector, int r, int c,int devnumber): rows{r}, cols{c} {
	assert(datavector.size() == rows*cols );
	int size=sizeof(float)*rows*cols;
	
	hipDeviceProp_t dev_prop;	
	hipGetDeviceProperties(&dev_prop,devnumber);
	assert(dev_prop.totalGlobalMem>size);

	gpuErrchk(hipMalloc( (void**)&d_data, size));
	gpuErrchk(hipMemcpy(d_data, datavector.data(), size, hipMemcpyHostToDevice));
}

void gMat::cleanup()
{
	gpuErrchk(hipFree(d_data));
}


//gMat::~gMat(){ 
	//	std::cout <<"de-allocating " << name << '\n' <<std::flush;	
	//	gpuErrchk(hipFree(d_data));
//}
int gMat::getrows(){ return rows;}
int gMat::getcols(){ return cols;}

std::vector<float> gMat::tovector(){
	float* h_data=new float[rows*cols];
	int size=rows*cols*sizeof(float);
	hipMemcpy(h_data,d_data,size,hipMemcpyDeviceToHost);
	std::vector<float> v;
	for (int i=0; i<rows; ++i) for (int j=0; j<cols; ++j){
		v.push_back(h_data[i*cols+j]);
	}
	delete[] h_data;
	return v;
}	

gMat randgMat(int r, int c){
	std::vector<float> datavector;
	std::default_random_engine re;
	std::uniform_real_distribution<float> dist{0.0,1.0};
	for (int i=0; i<r*c; i++) datavector.push_back(dist(re));
	return gMat(datavector,r,c);
}

gMat eye(int r){
	std::vector<float> datavector;
	for (int i=0; i!=r; ++i) for(int j=0; j!=r; ++j) 
		if(i==j) 
			datavector.push_back(1.0);
		else
			datavector.push_back(0.0);
	return gMat(datavector,r,r);
}

void prod(const gMat& A, const gMat& B, gMat& P, int mem){
	assert(A.cols==B.rows && A.rows==P.rows && B.cols==P.cols);
	int tile_size= floor(sqrt( mem/2/sizeof(float)));
    assert(tile_size>0);	
	
	matMulKernel<<<dim3(ceil(A.rows/ (float) tile_size), ceil( B.cols/ (float) tile_size)), dim3(tile_size,tile_size), mem>>>(A.d_data, B.d_data, P.d_data,A.rows, A.cols, B.cols, tile_size );
	gpuErrchk( hipPeekAtLastError() );	
}

std::ostream& operator<<(std::ostream& os,  gMat& gm)  
{  
	std::vector<float> v=gm.tovector();
	for (int i=0; i<gm.rows; ++i){
		for (int j=0; j<gm.cols; ++j){
			os << v[i*gm.cols+j] << ",";
		}
		os << "\n";
	}
	return os;  

}  


