#include "hip/hip_runtime.h"
#include <iostream>
const int TILE_SIZE=2;
//assume that A is m by n and B is n by s
//n is the inner dimension  shared by the two matrices
//A  and B will be broken into blocks of TILE_SIZE by TILE_SIZE
//P will be computed in blocks of TILE_SIZE by TILE_SIZE.  These blocks can be indexed by (b1,b2) where  0<=b1<m/TILE_SIZE and 0<=b2<n/TILE_SIZE.
//problem: if the matrix is not square, tile
__global__ void matMulKernel(float* A, float* B, float* P, int  n, int s){
	//each thread in the block will be responsible for a different element of these arrays
	__shared__ float Atile[TILE_SIZE][TILE_SIZE];
	__shared__ float Btile[TILE_SIZE][TILE_SIZE];
	
	int b1=blockIdx.x;
	int b2=blockIdx.y;
	int t1=threadIdx.x;
	int t2=threadIdx.y;

	int row=b1*TILE_SIZE+t1;// the indices of the element of P this thread is responsible for computing
	int col=b2*TILE_SIZE+t2;
	
	float Pvalue=0;
	//loop over the tiles necessary to compute the element of P of interest
	for (int j=0; j< n / TILE_SIZE;j++){
		//collaborate with other threads to store the current tiles of A and B
		//this thread is responsible for storing  the (t_1,t_2) element of the (b_1,j) tile of A and the (j,b_2) tile of B
		Atile[t1][t2]=A[ n*(b1*TILE_SIZE+t1) + j*TILE_SIZE  + t2 ]; // We access A[b1*TILE_SIZE+t1][j*TILE_SIZE+t2] using linearized indices (A has n columns)
		Btile[t1][t2]=B[ s*(j*TILE_SIZE+t1)  + b2*TILE_SIZE + t2 ]; //we access the B[j*TILE_SIZE+t1][b2*TILE_SIZE+t2] element using linearized indices (B has s columns)
		__syncthreads();

		//sum the elements of the current A and B tiles used in the computation of the element of P for which this thread is responsible
		for (int k=0; k<TILE_SIZE; k++){
			Pvalue += Atile[t1][k]*Btile[k][t2];
		}
		__syncthreads();

	}
	P[row*s + col]=Pvalue;

}
	

void printMat(float* A,int rows, int cols){
	for (int i=0; i<rows; ++i){
	for (int j=0; j< cols; ++j){
		std::cout << A[i*cols+j] << ",";
	}
		std::cout << "\n";
	}
	std::cout << std::endl;
}
void matMul(float* A, float* B, float* P, int m, int n, int s){		
	printMat(A,m,n);
	std::cout << "times \n";
	printMat(B,n,s);
	std::cout << " equals \n";
	float* d_A;
	float* d_B;
	float* d_P;
	hipMalloc ( (void**)&d_A, m*n*sizeof(float) );
	hipMalloc (  )
}
int main(){
	float A[]={1,2,3,4};
	float B[]={5,6,7,8};
	float P[4];
	

}
