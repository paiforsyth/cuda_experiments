
#include <hip/hip_runtime.h>
#include <iostream>
int main(){
	int dev_count;
	hipGetDeviceCount(&dev_count);
	hipDeviceProp_t dev_prop;
	for (int i=0; i<dev_count; i++){
		hipGetDeviceProperties(&dev_prop,i);
		std::cout << "Device number: " << i << "\n"; 
		std::cout << "Shared memory per block:" << dev_prop.sharedMemPerBlock  << "bytes \n";
		std::cout << "Global Memory:" << dev_prop.totalGlobalMem << "bytes";
	}
	std::cout<< std::flush;
}
