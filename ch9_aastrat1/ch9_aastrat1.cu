#include "hip/hip_runtime.h"
#include "gpuerrchk.cuh"
#include "real.h"

__global__ void ch9_aastrat1_kernel(char* buffer,unsigned int* histo, size_t inputsize){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int section_size = (inputsize-1) / (blockDim.x*gridDim.x) + 1;
	int start=i*section_size;


	for (int k=0; k< section_size; ++k){
		if (start+k < inputsize){
			int alphabet_position=buffer[start+k]-'a';
			if (alphabet_position >=0 && alphabet_position < 26)
				atomicAdd(&histo[alphabet_position/4], 1);
		}
	}
}
void ch9_aastrat1(char* buffer, unsigned int* histo,size_t inputsize){
	ch9_aastrat1_kernel<<<1, 512 >>>(buffer,histo,inputsize);
	gpuErrchk(hipPeekAtLastError());
}
