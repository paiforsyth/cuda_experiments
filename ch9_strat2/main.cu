#include "gpuerrchk.cuh"
#include "assert.h"
#include "real.h"
#include "ch9_strat2.cuh"
#include <iostream>
#include "rstring.h"
#include <string>
#include "hip/hip_runtime_api.h"
#define DAT_SIZE 1000000000


void test(){
	std::string buffer_string = rstring(DAT_SIZE);
    const char* buffer= buffer_string.c_str();	
	unsigned int histo[]= {0,0,0,0,0,0,0};
	 char* d_buffer;
	unsigned int* d_histo;
	gpuErrchk(hipMalloc((void**) &d_buffer,sizeof(char)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_buffer,buffer,sizeof(char)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_histo,sizeof(int)*7 ));
	
	ch9_strat2(d_buffer,d_histo, (size_t) DAT_SIZE);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(histo,d_histo,sizeof(int)*7,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_buffer));
	gpuErrchk(hipFree(d_histo));
}

int main(){
	test();
	std::cout<<"DONE!" <<std::endl;
	hipProfilerStop();
}

