#include "hip/hip_runtime.h"
#include "gpuerrchk.cuh"
#include "real.h"

__global__ void ch9_strat2_kernel(char* buffer,unsigned int* histo, size_t inputsize){

	unsigned int tid = threadIdx.x +blockIdx.x* blockDim.x;
	for (int i=tid; i< inputsize; i+= blockDim.x*gridDim.x  ){
		int alphabet_position=buffer[i]-'a';
		if (alphabet_position >=0 && alphabet_position < 26)
			atomicAdd(&histo[alphabet_position/4], 1);
	}
}
void ch9_strat2(char* buffer, unsigned int* histo,size_t inputsize){
	ch9_strat2_kernel<<<1, 512 >>>(buffer,histo,inputsize);
	gpuErrchk(hipPeekAtLastError());
}
