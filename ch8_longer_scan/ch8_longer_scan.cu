#include "hip/hip_runtime.h"
#include <math.h>
#include "gpuerrchk.cuh"
#include "real.h"
#include "next_pow_2.h"
#include <assert.h>
#include <iostream>
__global__ void ch8_longer_scan_kernel1(real* X, real* Y,real* S,unsigned int S_length, unsigned int treesize){
	extern __shared__ real XY[];
	int i=2*blockIdx.x*blockDim.x+threadIdx.x;
	 XY[threadIdx.x]=X[i];
	 XY[threadIdx.x+blockDim.x]=X[i+blockDim.x];	
	
	//up-sweep
	for (int stride=1; stride <= blockDim.x; stride*=2){
		__syncthreads();
		int index= 2*(threadIdx.x+1)*stride -1;
		if (index< treesize) XY[index]+=XY[index-stride];
	}


	//down-sweep
	for (int stride=treesize/2; stride>=1; stride/=2){
		__syncthreads();
		int index = 2*(threadIdx.x+1)*stride-1;
		if (index+stride < treesize){
			XY[index+stride]+=XY[index];
		}
	}
	__syncthreads();
	 Y[i]= XY[threadIdx.x];
	 Y[i+blockDim.x]= XY[threadIdx.x+blockDim.x]; 
	 if(threadIdx.x == blockDim.x-1)
		 S[blockIdx.x]= XY[treesize-1];
}

//performs an in-place scan on S
//full s length must be a power of 2
__global__ void ch8_longer_scan_kernel2(real* S, unsigned int full_S_length){
	extern __shared__ real XY[];
	int i=2*blockIdx.x*blockDim.x+threadIdx.x;
	XY[threadIdx.x]=S[i];
	XY[threadIdx.x+blockDim.x]=S[i+blockDim.x];	
	
	//up-sweep
	for (int stride=1; stride <= blockDim.x; stride*=2){
		__syncthreads();
		int index= 2*(threadIdx.x+1)*stride -1;
		if (index< full_S_length) XY[index]+=XY[index-stride];
	}


	//down-sweep
	for (int stride=full_S_length/2; stride>=1; stride/=2){
		__syncthreads();
		int index = 2*(threadIdx.x+1)*stride-1;
		if (index+stride < full_S_length){
			XY[index+stride]+=XY[index];
		}
	}
	__syncthreads();
	 S[i]= XY[threadIdx.x];
	 S[i+blockDim.x]=XY[threadIdx.x+blockDim.x]; 

}


__global__ void ch8_longer_scan_kernel3(real* Y, real* S){	
	int i=2*blockIdx.x*blockDim.x+threadIdx.x;
	if (blockIdx.x>0) {
		Y[i]+=S[blockIdx.x-1];
		Y[i+blockDim.x]+=S[blockIdx.x-1];
	}
}

//NEED TO DO MULTISTAGE SCAN ENTIRELY ON DEVICE, taking device input and writing device output

//treesize is assumed to be a power of 2.  d_X and d_Y are assumed to be of length length. 
//also assume treesize*(S_length)=length
//also assume d_S is small enough to be scanned by one thread block.
//also assume d_S points to an array with length equal to S_length rounded to the next power of 2
void ch8_longer_scan(real* d_X, real* d_Y,real* d_S, size_t length, unsigned int S_length, unsigned int treesize){
	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop,0); //assume we are using device 0
	size_t share_mem=dev_prop.sharedMemPerBlock;
	int thread_limit= dev_prop.maxThreadsPerBlock;
	size_t max_per_block=share_mem/sizeof(real);
	assert(treesize<=max_per_block && treesize<=2*thread_limit);
	assert(treesize*(S_length)==length);
	ch8_longer_scan_kernel1<<<S_length, treesize/2, treesize*sizeof(real)>>>(d_X, d_Y, d_S, S_length, treesize);
	gpuErrchk(hipPeekAtLastError());
	
	//debugging
//	real Y[2048];
//	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*2048,hipMemcpyDeviceToHost));
//	for(int i=0; i<2048; i++)
//		std::cout << "i=" << i << " Y[i]=" << Y[i] <<std::endl;




	unsigned int full_S_length=next_pow_2(S_length);
	assert(full_S_length <= max_per_block && full_S_length<=2*thread_limit);
	ch8_longer_scan_kernel2<<<1,full_S_length/2,sizeof(real)*full_S_length>>>(d_S,full_S_length);	
	gpuErrchk(hipPeekAtLastError());	
	ch8_longer_scan_kernel3<<<S_length,treesize/2, treesize*sizeof(real)>>>(d_Y,d_S);	
	gpuErrchk(hipPeekAtLastError());	
}
