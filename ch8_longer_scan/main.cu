
#include "gpuerrchk.cuh"
#include "assert.h"
#include "real.h"
#include "ch8_longer_scan.cuh"
#include <iostream>


void test2(){
	constexpr size_t DAT_SIZE=1024*1024;
	real* X= new real[DAT_SIZE];
	for (int i=0; i< DAT_SIZE; ++i) X[i]=1;
	real Y[DAT_SIZE];
	real* d_X;
	real* d_Y;
	real* d_S;
	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*DAT_SIZE ));	
	gpuErrchk(hipMalloc((void**) &d_S,sizeof(real)*DAT_SIZE ));
	ch8_longer_scan(d_X,d_Y,d_S,DAT_SIZE, DAT_SIZE/1024, 1024);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*DAT_SIZE,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));
	gpuErrchk(hipFree(d_S));
	delete[] X;	
}

int main(){
	test2();
}

