#include "gpuerrchk.cuh"
#include "assert.h"
#include "real.h"
#include "ch8_longer_scan.cuh"
#include <iostream>
void test(){
	constexpr size_t DAT_SIZE=1024*2;
	real X[DAT_SIZE];
	for (int i=0; i< DAT_SIZE; ++i) X[i]=1;
	real Y[DAT_SIZE];
	real* d_X;
	real* d_Y;
	real* d_S;
	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*DAT_SIZE ));	
	gpuErrchk(hipMalloc((void**) &d_S,sizeof(real)*DAT_SIZE ));
	ch8_longer_scan(d_X,d_Y,d_S,DAT_SIZE, DAT_SIZE/1024, 1024);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*DAT_SIZE,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));
	gpuErrchk(hipFree(d_S));
	for (int j=0; j<DAT_SIZE; ++j){
		//std::cout<< "j=" << j << " Y[j]=" << Y[j] <<std::endl;
		assert(Y[j]==j+1);
	}
}

void test2(){
	constexpr size_t DAT_SIZE=1024*100;
	real X[DAT_SIZE];
	for (int i=0; i< DAT_SIZE; ++i) X[i]=1;
	real Y[DAT_SIZE];
	real* d_X;
	real* d_Y;
	real* d_S;
	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*DAT_SIZE ));	
	gpuErrchk(hipMalloc((void**) &d_S,sizeof(real)*DAT_SIZE ));
	ch8_longer_scan(d_X,d_Y,d_S,DAT_SIZE, DAT_SIZE/1024, 1024);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*DAT_SIZE,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));
	gpuErrchk(hipFree(d_S));
	for (int j=0; j<DAT_SIZE; ++j){
		//std::cout<< "j=" << j << " Y[j]=" << Y[j] <<std::endl;
		assert(Y[j]==j+1);
	}
}

int main(){
	test();
	test2();
	std::cout<< "Success!!!" <<std::endl;
}

