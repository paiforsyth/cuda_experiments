#include "hip/hip_runtime.h"
#include "points.cuh"
#include <iostream>
__host__ __device__ Points::Points() : m_x(NULL), m_y(NULL){}
    
__host__ __device__ Points::Points(float* x, float* y): m_x(x), m_y(y){}

__host__ __device__  float2 Points::get_point(int idx) const{
    return make_float2(m_x[idx], m_y[idx]);
}

 __host__ void Points::print_point_d_2_h(int idx) const{
     //hack
   float* host_mx;
   host_mx=malloc(sizeof(float)*(idx+1) )
   hipMemcpy(host_mx, m_x,sizeof(float)*(idx+1), hipMemcpyDeviceToHost);
   float* host_my;
   host_my=malloc(sizeof(float)*(idx+1) )
   hipMemcpy(host_my, m_y,sizeof(float)*(idx+1), hipMemcpyDeviceToHost);
   std::cout << "x: " <<  host_mx[idx] << " y:" << host_my_[idx] << std::endl;
 }
 __host__ __device__  void Points::set_point(int idx, const float2& p){
    m_x[idx] = p.x;
    m_y[idx] = p.y; 
}
//set the actual pointers
__host__ __device__  void Points::set(float* x , float* y){
    m_x= x;
    m_y= y;
}
