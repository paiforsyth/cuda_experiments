#include "hip/hip_runtime.h"
#include "points.cuh"
#include <iostream>
__host__ __device__ Points::Points() : m_x(NULL), m_y(NULL){}
    
__host__ __device__ Points::Points(float* x, float* y): m_x(x), m_y(y){}

__host__ __device__ __forceinline__ float2 Points::get_point(int idx) const{
    return make_float2(m_x[idx], m_y[idx]);
}

 __host__ void Points::print_point(int idx) const{
   float2 p = get_point(idx);
   std::cout << "x: " <<  p.x << " y:" << p.y << std::endl;
 }
