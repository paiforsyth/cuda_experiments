#include "hip/hip_runtime.h"
#include "points.cuh"
__host__ __device__ Points::Points() : m_x(NULL), m_y(NULL){}
    
__host__ __device__ Points::Points(float* x, float* y): m_x(x), m_y(y){}

__host__ __device__ __forceinline__ float2 Points::get_point(int idx) const{
    return make_float2(m_x[idx], m_y[idx]);
}

