#include "hip/hip_runtime.h"
#include "points.cuh"
#include <iostream>
__host__ __device__ Points::Points() : m_x(NULL), m_y(NULL){}
    
__host__ __device__ Points::Points(float* x, float* y): m_x(x), m_y(y){}

__host__ __device__  float2 Points::get_point(int idx) const{
    return make_float2(m_x[idx], m_y[idx]);
}

 __host__ void Points::print_point_d_2_h(int idx) const{
     //hack
   float* host_mx;
   hipMemcpy(host_mx, mx,sizeof(float)*(idx+1), hipMemcpyDeviceToHost);
   float* host_my;
   hipMemcpy(host_my, my,sizeof(float)*(idx+1), hipMemcpyDeviceToHost);
   std::cout << "x: " <<  host_mx[idx] << " y:" << host_my_[idx] << std::endl;
 }
 __host__ __device__  void Points::set_point(int idx, const float2& p){
    m_x[idx] = p.x;
    m_y[idx] = p.y; 
}
//set the actual pointers
__host__ __device__  void Points::set(float* x , float* y){
    m_x= x;
    m_y= y;
}
