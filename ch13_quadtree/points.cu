#include "hip/hip_runtime.h"
#include "points.cuh"
#include <iostream>
__host__ __device__ Points::Points() : m_x(NULL), m_y(NULL){}
    
__host__ __device__ Points::Points(float* x, float* y): m_x(x), m_y(y){}

__host__ __device__ __forceinline__ float2 Points::get_point(int idx) const{
    return make_float2(m_x[idx], m_y[idx]);
}

 __host__ void Points::print_point(int idx) const{
   float2 p = get_point(idx);
   std::cout << "x: " <<  p.x << " y:" << p.y << std::endl;
 }
 __host__ __device__ __forceinline__ void set_point(int idx, const float2& p){
    m_x[idx] = p.x;
    m_y[idx] = p.y; 
}
//set the actual pointers
__host__ __device__ __forceinline__ void set(float* x , float* y){
    m_x= x;
    m_y= y;
}
