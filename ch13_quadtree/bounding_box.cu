#include "hip/hip_runtime.h"
#include "bounding_box.cuh"
__host__ __device__ BoundingBox::BoundingBox(){
    m_p_min = make_float2(-1.0f, -1.0f);
    m_p_max = make_float2(1.0f, 1.0f);
}

__host__ __device__ void BoundingBox::compute_center(float2 &center) const{
    center.x = (m_p_min.x + m_p_max.x)/2;
    center.y = (m_p_min.y + m_p_max.y )/2;    
}
__host__ __device__ const  float2& BoundingBox::get_max() const{
        return m_p_max;    
}
__host__ __device__ const float2& BoundingBox::get_min() const{
    return m_p_min;    
}
__host__ __device__ bool BoundingBox::contains(float2& p) const{
    return p.x >= m_p_min.x && p.y >= m_p_min.y && p.x <= m_p_max.x && p.y <= m_p_max.y;
}

__host__ __device__ void BoundingBox::set(float min_x, float min_y, float max_x, float max_y){
    m_p_min.x = min_x;
    m_p_min.y = min_y;
    m_p_max.x = max_x;
    m_p_max.y = max_y;
}


