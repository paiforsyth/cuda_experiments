#include "hip/hip_runtime.h"
#include <iostream>
#include "bounding_box.cuh"
int main(){
    BoundingBox box;        
    float2 p1 = make_float2(0.5f,0.5f);
    float2 p2 = make_float2(10.0f, 10.0f);

    std::cout << "Does point 1 lie in the box? " << box.contains(p1) << std::endl;
    std::cout << "Does point 2 lie in the box? " <<< box.contains(p2) << std::endl;

}
