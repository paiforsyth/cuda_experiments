#include "hip/hip_runtime.h"
#include <iostream>
int main(){
    BoundingBox box;        
    p1 = make_float2(0.5f,0.5f);
    p2 = make_float2(10.0f, 10.0f);

    std::cout << "Does point 1 lie in the box? " << box.contains(p1) << std::endl;
    std::cout << "Does point 2 lie in the box? " <<< box.contains(p2) << std::endl;

}
