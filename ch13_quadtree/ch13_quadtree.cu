#include "hip/hip_runtime.h"
#include "gpuerrchk.cuh"
#include "real.h"
#include "quad_tree_node.cuh" 
#include "points.cuh"
#include "parameters.cuh"
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <random>
#include <cstdlib>
#include <iostream>
#include <assert.h>
#include <stdio.h>
#define SECTION_SIZE 512
__global__ void ch13_quadtree_kernel(real* X, real* Y, int inputsize){
}

void ch13_quadtree(real* d_X, real* d_Y,int inputsize){
	ch13_quadtree_kernel<<<ceil(inputsize/ (real) SECTION_SIZE),SECTION_SIZE>>>(d_X,d_Y,inputsize);
	gpuErrchk(hipPeekAtLastError());
}




//currenty, my interpretation is the following:
//there are two points buffers, which alternate being active.  Within the currently active points buffer, the points associated with a particular node are in the range [node.points_begin(), node.points_end()).  These points are split amoung the various threads, whith one block of threads being assigned to each node.  thread i handles points with indexes starting from nodes.points_begin()+threadIdx.x and increase at intervals of Blockdim.x
__device__ bool check_num_points_and_depth(QuadTreeNode& node, Points* points, int num_points, Parameters params){
    if (params.depth == params.max_depth || num_points <= params.min_points_per_node){
        //stop the recurrsion, making sure points[0] contains all the points
        if (params.point_selector == 1){
            int it = node.points_begin();
            int end=node.points_end();
            for (it += threadIdx.x; it < end; it+=blockDim.x)
                if (it< end)
                    points[0].set_point(it, points[1].get_point(it));
        }
        return true;
    
    }
    return false;
}


//count the number of points in each quadrant
__device__ void count_points_in_children(const Points& in_points, int* smem, int range_begin, int range_end, float2 center){
     if(threadIdx.x < 4) smem[threadIdx.x] = 0;
     __syncthreads();
     //count the points in each quadraant, with each thread dealing with its own points
    for(int iter=range_begin+threadIdx.x; iter< range_end; iter+=blockDim.x){
        float2 p = in_points.get_point(iter);
        if(p.x < center.x && p.y >= center.y) //top left
           atomicAdd(&smem[0],1);
        if(p.x>= center.x && p.y >= center.y) //top right
           atomicAdd(&smem[1],1);
        if(p.x < center.x && p.y < center.y) //bottom left
            atomicAdd(&smem[2],1);
        if(p.x >= center.x && p.y <center.y) //bottom right
            atomicAdd(&smem[3],1);
    }
    __syncthreads();

}

__device__ void scan_for_offsets(int node_points_begin, int* smem){
    int* smem2 = &smem[4];
    if (threadIdx.x ==0 ){
        for(int i = 0; i<4; i++)
            smem2[i] = i==0 ? 0 : smem2[i-1] + smem[i-1];
        for(int i = 0; i<4; i++)
            smem2[i]+=node_points_begin;
    
    }
    __syncthreads();
}

//reorder points to group those in the same quadrant
//smem2[i] starts holding the number of points in quadrants ordered before
//quadrant i, and ends up recording the number of points in quadrants before or including quadrant i
__device__ void reorder_points(Points& out_points, const Points& in_points, int* smem, int range_begin, int range_end, float2 center){
    int* smem2 = &smem[4];
    for(int iter =range_begin +threadIdx.x; iter<range_end; iter+=blockDim.x){
        int dest;
        float2 p =in_points.get_point(iter);
        if(p.x < center.x && p.y >= center.y) //top left
           dest= atomicAdd(&smem2[0],1);
        if(p.x>= center.x && p.y >= center.y) //top right
           dest= atomicAdd(&smem2[1],1);
        if(p.x < center.x && p.y < center.y) //bottom left
           dest= atomicAdd(&smem2[2],1);
        if(p.x >= center.x && p.y <center.y) //bottom right
           dest= atomicAdd(&smem2[3],1);
        out_points.set_point(dest,p);
    }
    __syncthreads();
}

__device__ void prepare_children(QuadTreeNode* children, QuadTreeNode& node, const BoundingBox& bbox, int* smem, bool* active_nodes){
    //int child_offset = 4*node.id();  
    int child_offset = node.id();//I added this, since the above seemed to be wrong
    children[child_offset+0].set_id(4*node.id()+0);
    children[child_offset+1].set_id(4*node.id()+4);
    children[child_offset+2].set_id(4*node.id()+8);
    children[child_offset+3].set_id(4*node.id()+12);

    active_nodes[child_offset+0]=true;
    active_nodes[child_offset+1]=true;
    active_nodes[child_offset+2]=true;
    active_nodes[child_offset+3]=true;


    //points in bounding box:
    const float2& pmin = bbox.get_min();
    const float2& pmax = bbox.get_max();
    float2 center;
    bbox.compute_center(center); //does this work, given that bbox is const?

    children[child_offset+0].set_bounding_box(pmin.x, center.y, center.x, pmax.y ); //top left
    children[child_offset+1].set_bounding_box(center.x, center.y, pmax.x, pmax.y ); //top right
    children[child_offset+2].set_bounding_box(pmin.x, pmin.y, center.x, center.y ); //bottom left
    children[child_offset+3].set_bounding_box(center.x, pmin.y, pmax.x, center.y ); //bottom right

    //set the point ranges for the children
    children[child_offset + 0].set_range(node.points_begin(), smem[4 + 0]);
    children[child_offset + 1].set_range(smem[4 + 0], smem[4 + 1 ]);
    children[child_offset + 2].set_range(smem[4 + 1], smem[4 + 2 ]);
    children[child_offset + 3].set_range(smem[4 + 2], smem[4 + 3 ]);
    
}


__global__ void build_quad_tree_kernel(QuadTreeNode* nodes, Points* points, Parameters params,bool* active_nodes  ){
    __shared__ int smem[8];
    //idea: have only thread 0 display the point
    
    //the current node
    QuadTreeNode& node = nodes[blockIdx.x]; 
    node.set_id(node.id() + blockIdx.x);
    int num_points = node.num_points();
    //printf("hello!");
    hipDeviceSynchronize();
    if(threadIdx.x==0){
       // printf("yo");
        printf("Depth: %d.  (depth-specific) node num: %d.  Point range: %d to %d. bbox : min= (%f,%f) max=(%f,%f).    \n",  params.depth, blockIdx.x, node.points_begin(), node.points_end(), node.bounding_box().get_min().x, node.bounding_box().get_min().y, node.bounding_box().get_max().x, node.bounding_box().get_max().y    );
    }
    hipDeviceSynchronize();

    //check exit condition, moving points to first buffer as needed
    bool exit = check_num_points_and_depth(node, points, num_points, params);
    if(exit) return;
    
    const BoundingBox& bbox = node.bounding_box();
    float2 center;
    bbox.compute_center(center); //does this work, given that bbox is const?

    int range_begin = node.points_begin();
    int range_end = node.points_end();
    const Points& in_points = points[params.point_selector];
    Points& out_points = points[(params.point_selector +1) %2];

    //count points in each child
    count_points_in_children(in_points, smem, range_begin, range_end, center);
    if (threadIdx.x == 0){
        printf("Depth: %d.  Num: %d.  Points in children: (%d,%d,%d,%d) \n", params.depth, blockIdx.x, smem[0], smem[1], smem[2], smem[3] );
    }
    //compute reordering offset for each quadrant
    scan_for_offsets(node.points_begin(), smem);
    

    //reorder points (in other point buffer)
    reorder_points(out_points, in_points, smem, range_begin, range_end, center);

    
    if (threadIdx.x == blockDim.x-1){
        printf("Depth: %d.  Num: %d launching child kernels\n", params.depth, blockIdx.x);
        QuadTreeNode* children = &nodes[params.num_nodes_at_this_level];
        bool* child_active_nodes = &active_nodes[params.num_nodes_at_this_level];
        prepare_children(children, node, bbox, smem,child_active_nodes);
        //launch child kernels
        build_quad_tree_kernel<<<4,blockDim.x, 8 * sizeof(int)>>>(children, points, Parameters(params, true), child_active_nodes);
    }



}

class RNG_functor{
    int ip;
    public:
        RNG_functor(int init_p){  
            ip = init_p; 
        }

    __host__ __device__
    float operator() (int dex) const 
        {
            thrust::default_random_engine generator;
            thrust::uniform_real_distribution<float> distribution(-1.0f,1.0f);
            generator.discard(ip+dex);
            return distribution(generator);
        }
};

int main(int argc, char **argv){
    std::cout << "Recieved " << argc << " arguments."<<std::endl;
    assert(argc == 4);
    //load paramters from command line
    const int num_points = atoi(argv[1]);
    const int max_depth = atoi(argv[2]);   
    const int min_points_per_node = atoi(argv[3]);
    std::cout << "Number of points is: "  << num_points << std::endl;
    std::cout << "Max depth is: "  << max_depth << std::endl;
    std::cout << "Min points per node is "  << min_points_per_node << std::endl;
    

    //allocate memory for points 
    hipDeviceSynchronize();
    std::cout << "allocating device memory for points " << std::endl;
    thrust::device_vector<float> x_d0(num_points);
    thrust::device_vector<float> x_d1(num_points);
    thrust::device_vector<float> y_d0(num_points);
    thrust::device_vector<float> y_d1(num_points);

    //generate random points
    hipDeviceSynchronize();
    std::cout << "generating random points " << std::endl;
    thrust::counting_iterator<int> count1_begin(0);
    thrust::counting_iterator<int> count2_begin(0);
    thrust::transform(count1_begin, count1_begin+num_points,x_d0.begin(),RNG_functor(0) );
    thrust::transform(count2_begin, count2_begin+num_points,y_d0.begin(),RNG_functor(42) );

    /*thrust::generate(
            thrust::make_zip_iterator(thrust::make_tuple(x_d0.begin(),y_d0.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(x_d0.end(), y_d0.end())),
            rng
            );
            */
    

     // host Points object whose uderlying data is the key  device_vectors
    hipDeviceSynchronize();
    std::cout << "setting up points_init " << std::endl;
    hipDeviceSynchronize();
    Points points_init[2];
    points_init[0].set(thrust::raw_pointer_cast(&x_d0[0]),
                      thrust::raw_pointer_cast(&y_d0[0]) 
            );
    points_init[1].set(thrust::raw_pointer_cast(&x_d1[0]),
             thrust::raw_pointer_cast(&y_d1[0]) 
            );

    //allocate Points objects on the device, refering to the same underlying data as above
    hipDeviceSynchronize();
    std::cout << "allocating points object on device " << std::endl;
    hipDeviceSynchronize();
    Points* points; 
    hipMalloc( (void**) &points, 2*sizeof(Points) );
    hipMemcpy(points, points_init, 2*sizeof(Points), hipMemcpyHostToDevice);
    
    //count the maximum number of nodes that could be needed
    int max_nodes = 0;
    for (int i=0, num_nodes_at_level=1; i<max_depth;++i, num_nodes_at_level*=4 )
        max_nodes += num_nodes_at_level;

    //alocate memory to store the tree
    hipDeviceSynchronize();
    std::cout << "allocating tree " << std::endl;
    hipDeviceSynchronize();
    QuadTreeNode root;
    root.set_range(0, num_points);
    QuadTreeNode* nodes;
    hipMalloc((void**) &nodes, max_nodes*sizeof(QuadTreeNode) );
    hipMemcpy(nodes, &root, sizeof(QuadTreeNode), hipMemcpyHostToDevice );


    hipDeviceSynchronize();
    std::cout << "allocating host_active_nodes " << std::endl;
    hipDeviceSynchronize();
    bool* host_active_nodes = new bool[max_nodes];
    host_active_nodes[0]=true;
    for (int i=1; i< max_nodes; i++)
        host_active_nodes[i]=false;

    hipDeviceSynchronize();
    std::cout << "allocating device active nodes " << std::endl;
    hipDeviceSynchronize();
    bool* device_active_nodes;
    hipMalloc((void**) &device_active_nodes, max_nodes*sizeof(bool)  );
    hipDeviceSynchronize();
    std::cout << "copying host active nodes to device active nodes " << std::endl;
    hipDeviceSynchronize();
    hipMemcpy(device_active_nodes, &host_active_nodes[0], sizeof(bool)*max_nodes, hipMemcpyHostToDevice);
    
    //set reucsion limit for cuda dynamic parallelism to max_depth
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth,max_depth);

    //build the tree
    Parameters params(max_depth, min_points_per_node);
    const int NUM_THREADS_PER_BLOCK=32;
    const size_t smem_size=8*sizeof(int);
    std::cout << "Launching Kernel"<<std::endl;
    build_quad_tree_kernel<<<1, NUM_THREADS_PER_BLOCK,smem_size>>>(nodes, points, params, device_active_nodes);
	gpuErrchk(hipPeekAtLastError());


    //inspect results
    //copy the nodes
    QuadTreeNode* host_nodes;
    host_nodes = (QuadTreeNode*) malloc( max_nodes * sizeof(QuadTreeNode) );
    hipMemcpy(host_nodes,&nodes, max_nodes*sizeof(QuadTreeNode) , hipMemcpyDeviceToHost);

    //copy the bools
    hipMemcpy(host_active_nodes, &device_active_nodes, sizeof(bool)*max_nodes, hipMemcpyDeviceToHost);

    

    for(int i=0; i<max_nodes; ++i)
        if(host_active_nodes[i]){
            std::cout<< "Printing Node number: "<< i << std::endl;
            //host_nodes[i].list_points(points[0]);
        }
    //free memory
    hipFree(nodes);
    hipFree(points);
}
