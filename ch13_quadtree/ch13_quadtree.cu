#include "hip/hip_runtime.h"
#include "gpuerrchk.cuh"
#include "real.h"
#define SECTION_SIZE 512
__global__ void ch13_quadtree_kernel(real* X, real* Y, int inputsize){
}

void ch13_quadtree(real* d_X, real* d_Y,int inputsize){
	ch13_quadtree_kernel<<<ceil(inputsize/ (real) SECTION_SIZE),SECTION_SIZE>>>(d_X,d_Y,inputsize);
	gpuErrchk(hipPeekAtLastError());
}
