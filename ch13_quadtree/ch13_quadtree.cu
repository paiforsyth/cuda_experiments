#include "hip/hip_runtime.h"
#include "gpuerrchk.cuh"
#include "real.h"
#include "quad_tree_node.cuh" 
#include "points.cuh"
#include "parameters.cuh"
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_vector.h>
#include <random>
#include <cstdlib>

#define SECTION_SIZE 512
__global__ void ch13_quadtree_kernel(real* X, real* Y, int inputsize){
}

void ch13_quadtree(real* d_X, real* d_Y,int inputsize){
	ch13_quadtree_kernel<<<ceil(inputsize/ (real) SECTION_SIZE),SECTION_SIZE>>>(d_X,d_Y,inputsize);
	gpuErrchk(hipPeekAtLastError());
}




//currenty, my interpretation is the following:
//there are two points buffers, which alternate being active.  Within the currently active points buffer, the points associated with a particular node are in the range [node.points_begin(), node.points_end()).  These points are split amoung the various threads, whith one block of threads being assigned to each node.  thread i handles points with indexes starting from nodes.points_begin()+threadIdx.x and increase at intervals of Blockdim.x
__device__ bool check_num_points_and_depth(QuadTreeNode& node, Points* points, int num_points, Parameters params){
    if (params.depth == params.max_depth || num_points <= params.min_points_per_node){
        //stop the recurrsion, making sure points[0] contains all the points
        if (params.point_selector == 1){
            int it = node.points_begin();
            int end=node.points_end();
            for (it += threadIdx.x; it < end; it+=Blockdim.x)
                if (it< end)
                    points[0].set_point(it, points[1].get_point(it));
        }
        return true;
    
    }
    return false;
}


//count the number of points in each quadrant
__device__ void count_points_in_children(const Points& in_points, int* smem, int range_begin, int range_end, float2 center){
     if(threadIdx.x < 4) smem[threadIdx.x] = 0;
     __syncthreads();
     //count the points in each quadraant, with each thread dealing with its own points
    for(int iter=range_begin+threadIdx.x; iter< range_end; iter+=blockDim.x){
        float2 p = in_points.get_point(iter);
        if(p.x < center.x && p.y >= center.y) //top left
           atomicAdd(&smem[0],1);
        if(p.x>= center.x && p.y >= center.y) //top right
           atomicAdd(&smem[1],1);
        if(p.x < center.x && p.y < center.y) //bottom left
            atomicAdd(&smem[2],1);
        if(p.x >= center.x && p.y <center.y) //bottom right
            atomicAdd(&smem[3],1);
    }
    __syncthreads();

}

__device__ void scan_for_offsets(int node_points_begin, int* smem){
    int* smem2 = &smem[4];
    if (threadIdx.x ==0 ){
        for(int i = 0; i<4; i++)
            smem2[i] = i==0 ? 0 : smem2[i-1] + smem[i-1];
        for(int i = 0; i<4; i++)
            smem2[i]+=node_points_begin;
    
    }
    __syncthreads();
}

//reorder points to group those in the same quadrant
//smem2[i] starts holding the number of points in quadrants ordered before
//quadrant i, and ends up recording the number of points in quadrants before or including quadrant i
__device__ void reorder_points(Points& out_points, const Points& in_points, int* smem, int range_begin, int range_end, float2 center){
    int* smem2 = &smem[4];
    for(int iter =range_begin +threadIdx.x; iter<range_end; iter+=blockDim.x){
        int dest;
        float2 p =in_points.get_point(iter);
        if(p.x < center.x && p.y >= center.y) //top left
           dest= atomicAdd(&smem2[0],1);
        if(p.x>= center.x && p.y >= center.y) //top right
           dest= atomicAdd(&smem2[1],1);
        if(p.x < center.x && p.y < center.y) //bottom left
           dest= atomicAdd(&smem2[2],1);
        if(p.x >= center.x && p.y <center.y) //bottom right
           dest= atomicAdd(&smem2[3],1);
        out_points.set_point(dest,p);
    }
    __syncthreads();
}

__device__ prepare_children(QuadTreeNode* children, QuadTreeNode& node, const BoundingBox& bbox, int* smem){
    int child_offset = 4*node.id();
    children[child_offset+0].set_id(4*node.id()+0);
    children[child_offset+1].set_id(4*node.id()+4);
    children[child_offset+2].set_id(4*node.id()+8);
    children[child_offset+3].set_id(4*node.id()+12);

    //points in bounding box:
    const float2& pmin = bbox.get_min();
    const float2& pmax = bbox.get_max();

    children[child_offset+0].set_bounding_box(pmin.x, center.y, center.x, pmax.y ) //top left
    children[child_offset+1].set_bounding_box(center.x, center.y, pmax.x, pmax.y ) //top right
    children[child_offset+2].set_bounding_box(pmin.x, pmin.y, center.x, center.y ) //bottom left
    children[child_offset+3].set_bounding_box(center.x, pmin.y, pmax.x, center.y ) //bottom right

    //set the point ranges for the children
    children[child_offset + 0].set_range(node.points_begin(0), smem[4 + 0]);
    children[child_offset + 1].set_range(smem[4 + 0], smem[4 + 1 ]);
    children[child_offset + 2].set_range(smem[4 + 1], smem[4 + 2 ]);
    children[child_offset + 3].set_range(smem[4 + 2], smem[4 + 3 ]);
    
}


__global__ void build_quad_tree_kernel(QuadTreeNode* nodes, Points* points, Parameters params  ){
    __shared__ int smem[8];

    //the current node
    QuadTreeNode& node = nodes[blockIdx.x];
    node.set_idx(node.id() + blockIdx.x);
    int num_points = node.num_points();

    //check exit condition, moving points to first buffer as needed
    bool exit = check_num_points_and_depth(node, points, num_points, params);
    if(exit) return;
    
    const BoundingBox& bbox = node.bounding_box();
    float2 center;
    bbox.compute_center(center); //does this work, given that bbox is const?

    int range_begin = node.points_begin();
    int range_end = node.points_end();
    const Points& in_points = points[params.point_selector];
    Points& out_points = points[(params.point_selector +1) %2];

    //count points in each child
    count_points_in_children(in_pointsl, smem, range_begin, range_endm cebter);
    //compute reordering offset for each quadrant
    scan_for_offsets(node.points_begin(), smem);
    

    //reorder points (in other point buffer)
    reoder_points(out_points, in_points, smem, range_begin, range_end, center);

    
    if (threadIdx.x == blockDim.x-1){
        QuadTreeNode* children = &nodes[params.num_nodes_at_this_level];

        prepare_children(children, node, bbox, smem);
        //launch child kernels
        build_quad_tree_kernel<<4,blockDim.x, 8 * sizeof(int)>>(children, points, Parameters(params, true));
    }



}

void main(int argc, char **argv){
    //load paramters from command line
    const int num_points = atoi(argv[0]);
    const int max_depth = atoi(argv[1]);   
    const int min_points_per_node = atoi(argv[2]);
    

    //allocate memory for points
    thrust::device_vector<float> x_d0(num_points);
    thrust::device_vector<float> x_d1(num_points);
    thrust::device_vector<float> y_d0(num_points);
    thrust::device_vector<float> y_d1(num_points);

    //generate random points
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(-1.0,1.0);
    rng=[&](){return distribution(generator);}
    thrust::generate(
            thrust::make_zip_iterator(thrust::make_tuple(x_d0.begin(),y_d0.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(x_d0.end(), y_d0.end())),
            rng
            );
     // host Points object pointing to the key device_vectors
    Point points_init[2];
    points_init[0].set(thrust::raw_pointer_cast(&x_d0[0])
                      thrust::raw_pointer_cast(&y_d0[0]) 
            );
    points_init[1].set(thrust::raw_pointer_cast(&x_d1[0]),
             thrust::raw_pointer_cast(&y_d1[0]) 
            );

    //allocate Points objects on the device, refering to the same underlying data as above
    Points* points; 
    hipMalloc( (void**) &points, 2*sizeof(Points) );
    hipMemcpy(points, points_init, 2*sizeof(Points), hipMemcpyHostToDevice);
    
    //count the maximum number of nodes that could be needed
    int max_nodes = 0;
    for (int i=0, num_nodes_at_level=1; i<max_depth;++i, num_nodes_at_level*=4 )
        max_nodes += num_nodes_at_level

    //alocate memory to store the tree
    QuadTreeNode root;
    root.set_range(0, num_points);
    QuadTreeNode* nodes;
    hipMalloc((void**) &nodes, max_nodes*sizeof(QuadTreeNode) );
    hipMemcpy(nodes, &root, sizeof(QuadTreeNode), hipMemcpyHostToDevice );
    
    //set reucsion limit for cuda dynamic parallelism to max_depth
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth,max_depth);

    //build the tree
    Parameters parameters(max_depth, min_points_per_node);
    const int NUM_THREADS_PER_BLOCK=32;
    const size_t smem_size=8*sizeof(int);
    build_quad_tree_kernel<<<1, NUM_THREADS_PER_BLOCK,smem_size>>>(nodes, points, params);
	gpuErrchk(hipPeekAtLastError());

    //free memory
    hipFree(nodes);
    hipFree(points);
