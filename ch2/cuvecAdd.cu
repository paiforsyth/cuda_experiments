#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
//compute  vector sum C=A+B
//each thread performs one pair-wise addition.
__global__
void vecAddKernel(float* A, float* B, float* C, int n){
	int i=blockDim.x*blockIdx.x + threadIdx.x;
	if (i<n) C[i]= A[i]+ B[i];
}
void vecAdd(float* A, float* B, float* C,int n){
	int size=n*sizeof(float);
	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc( (void**)&d_A, size);
	hipMalloc( (void**)&d_B, size);
	hipMalloc( (void**)&d_C, size);
	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,A,size,hipMemcpyHostToDevice);
	vecAddKernel<<<ceil(n/256.0), 256>>>(d_A,d_B,d_C,n);
	
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
	hipFree(d_A);
