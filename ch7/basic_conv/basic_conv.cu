#include "hip/hip_runtime.h"
#include "real.h"
#include "gpuerrchk.cuh"
#include "math.h"


__global__ void basic_conv_kernel(real* A, real* M, real* P, int mask_width, int width){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	real Pvalue=0; //mask width is assumed odd  So there are mask_width values in [-mask_width/2, mask_width/2]
	for(int j=i-mask_width/2; j<=i+mask_width/2; ++j){
		if (j>=0 && j<width)
			Pvalue+= A[j]*M[j-(i-mask_width/2)];

	}
	P[i]=Pvalue;
}

void basic_conv(real* A, real* M, real* P, int mask_width, int width ){
	real* d_A;
	real* d_M;
	real* d_P;
	gpuErrchk(hipMalloc((void**)&d_A, sizeof(real)*width ));
	gpuErrchk(hipMemcpy(d_A, A, sizeof(real)*width, hipMemcpyHostToDevice )  );
	gpuErrchk(hipMalloc((void**)&d_M, sizeof(real)*mask_width ));
	gpuErrchk(hipMemcpy(d_M, M, sizeof(real)*mask_width, hipMemcpyHostToDevice )  );
	gpuErrchk(hipMalloc((void**)&d_P, sizeof(real)*width ));
	int blocksize=512;
	basic_conv_kernel<<<ceil(width/ (real)blocksize),blocksize >>>(d_A, d_M, d_P, mask_width, width);
	gpuErrchk(hipMemcpy(P, d_P, sizeof(real)*width, hipMemcpyDeviceToHost )  );
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipFree(d_A ) );
	gpuErrchk(hipFree(d_M ) );
	gpuErrchk(hipFree(d_P ) );

}
