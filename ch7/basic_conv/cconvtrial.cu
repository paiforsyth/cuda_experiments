#include "hip/hip_runtime.h"
#include "assert.h"
#include "real.h"
#include <iostream>
#include "gpuerrchk.cuh"
#include "math.h"

#define MAX_MASK_WIDTH 10
__device__ __constant__ float d_M[1000];

__global__ void constant_conv_kernel(real* A, real* P, int mask_width, int width){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	real Pvalue=0; //mask width is assumed odd  So there are mask_width values in [-mask_width/2, mask_width/2]
	for(int j=i-mask_width/2; j<=i+mask_width/2; ++j){
		if (j>=0 && j<width)
			Pvalue+= A[j]*d_M[j-(i-mask_width/2)];

	}
	P[i]=Pvalue;
}

void constant_conv(real* A,float* M, real* P, int mask_width, int width ){
	real* d_A;
	real* d_P;
	gpuErrchk(hipMalloc((void**)&d_A, sizeof(real)*width ));
	gpuErrchk(hipMemcpy(d_A, A, sizeof(real)*width, hipMemcpyHostToDevice )  );
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_M), M, sizeof(real)*mask_width )  );
	gpuErrchk(hipMalloc((void**)&d_P, sizeof(real)*width ));
	int blocksize=512;
	constant_conv_kernel<<<ceil(width/ (real)blocksize),blocksize >>>(d_A,  d_P, mask_width, width);
	gpuErrchk(hipMemcpy(P, d_P, sizeof(real)*width, hipMemcpyDeviceToHost )  );
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipFree(d_A ) );
	gpuErrchk(hipFree(d_P ) );

}
void trial(){
	constexpr int asize=10^5;
	constexpr int bsize=1000;
	real A[asize];
	for(int i=0; i< asize; i++){
		A[i]=1;
	}
	real M[bsize];
	for (int i=0; i<bsize; ++i){
		M[i]=i;	
	}
	real P[asize];
	constant_conv(A,M,P,bsize,asize);
}

int main(){
	trial();
	
}
