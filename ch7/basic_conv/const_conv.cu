#include "hip/hip_runtime.h"
#include "real.h"
#include "gpuerrchk.cuh"
#include "math.h"

#define MAX_MASK_WIDTH 10
__device__ __constant__ float d_M[MAX_MASK_WIDTH];

__global__ void constant_conv_kernel(real* A, real* P, int mask_width, int width){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	real Pvalue=0; //mask width is assumed odd  So there are mask_width values in [-mask_width/2, mask_width/2]
	for(int j=i-mask_width/2; j<=i+mask_width/2; ++j){
		if (j>=0 && j<width)
			Pvalue+= A[j]*d_M[j-(i-mask_width/2)];

	}
	P[i]=Pvalue;
}

void constant_conv(real* A,float* M, real* P, int mask_width, int width ){
	real* d_A;
	real* d_M;
	real* d_P;
	gpuErrchk(hipMalloc((void**)&d_A, sizeof(real)*width ));
	gpuErrchk(hipMemcpy(d_A, A, sizeof(real)*width, hipMemcpyHostToDevice )  );
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_M), M, sizeof(real)*mask_width )  );
	gpuErrchk(hipMalloc((void**)&d_P, sizeof(real)*width ));
	int blocksize=512;
	constant_conv_kernel<<<ceil(width/ (real)blocksize),blocksize >>>(d_A,  d_P, mask_width, width);
	gpuErrchk(hipMemcpy(P, d_P, sizeof(real)*width, hipMemcpyDeviceToHost )  );
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipFree(d_A ) );
	gpuErrchk(hipFree(d_M ) );
	gpuErrchk(hipFree(d_P ) );

}
