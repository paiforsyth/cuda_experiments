#include "hip/hip_runtime.h"
#include "assert.h"
#include "real.h"
#include <iostream>
#include "gpuerrchk.cuh"
#include "math.h"

#define MAX_MASK_WIDTH 10
#define TILE_SIZE 1000
__device__ __constant__ float d_M[1000];

__global__ void share_conv_kernel(real* A, real* P, int mask_width, int width){
	__shared__ real A_s[TILE_SIZE];	
	A_s[threadIdx.x]=A[blockIdx.x*blockDim.x+threadIdx.x];
	__syncthreads();	
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int this_tile_start_point = blockIdx.x*blockDim.x;
	int next_tile_start_point = (blockIdx.x+1)*blockDim.x;
	int mask_start_point= i-mask_width/2;	
	real Pvalue=0; //mask width is assumed odd  So there are mask_width integers in [-mask_width/2, mask_width/2]
	for (int j=0; j< mask_width; j++){
		int mask_index= mask_start_point +j;
		if( mask_index< 0  || mask_index >= width)
			continue;
		if(	mask_index >=this_tile_start_point && mask_index < next_tile_start_point)
			Pvalue+=A_s[threadIdx.x+j -mask_width/2]*d_M[j];
		else
			Pvalue+=A[mask_index]*d_M[j];
	}
	P[i]=Pvalue;
}

void share_conv(real* A,float* M, real* P, int mask_width, int width ){
	real* d_A;
	real* d_P;
	gpuErrchk(hipMalloc((void**)&d_A, sizeof(real)*width ));
	gpuErrchk(hipMemcpy(d_A, A, sizeof(real)*width, hipMemcpyHostToDevice )  );
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_M), M, sizeof(real)*mask_width )  );
	gpuErrchk(hipMalloc((void**)&d_P, sizeof(real)*width ));
	int blocksize=512;
	share_conv_kernel<<<ceil(width/ (real)blocksize),blocksize >>>(d_A,  d_P, mask_width, width);
	gpuErrchk(hipMemcpy(P, d_P, sizeof(real)*width, hipMemcpyDeviceToHost )  );
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipFree(d_A ) );
	gpuErrchk(hipFree(d_P ) );
}

/*void trial(){
	constexpr int asize=10^5;
	constexpr int bsize=1000;
	real A[asize];
	for(int i=0; i< asize; i++){
		A[i]=1;
	}
	real M[bsize];
	for (int i=0; i<bsize; ++i){
		M[i]=i;	
	}
	real P[asize];
	share_conv(A,M,P,bsize,asize);
}
*/


	

