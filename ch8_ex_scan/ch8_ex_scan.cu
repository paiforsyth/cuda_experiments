#include "hip/hip_runtime.h"
#include "gpuerrchk.cuh"
#include "real.h"
#define SECTION_SIZE 1024 

//input size=SECTION_SIZE is twice block size, so that we can have 2 threads per element to be summed
__global__ void ch8_ex_scan_kernel(real* X, real* Y, int inputsize){
	__shared__ real XY[SECTION_SIZE];
	int i=2*blockIdx.x*blockDim.x+threadIdx.x;
	if (i < inputsize) XY[threadIdx.x]=X[i];
	if (i + blockDim.x <inputsize) XY[threadIdx.x+blockDim.x]=X[i+blockDim.x];	
	
	//up-sweep
	for (int stride=1; stride <= blockDim.x; stride*=2){
		__syncthreads();
		int index= 2*(threadIdx.x+1)*stride -1;
		if (index< SECTION_SIZE) XY[index]+=XY[index-stride];
	}


	//down-sweep
	for (int stride=SECTION_SIZE/2; stride>=1; stride/=2){
		__syncthreads();
		int index = 2*(threadIdx.x+1)*stride-1;
		if (index+stride < SECTION_SIZE){
			XY[index+stride]+=XY[index];
		}
	}
	__syncthreads();
	if (i< inputsize) Y[i]= XY[threadIdx.x];
	if (i+blockDim.x < inputsize) Y[i+blockDim.x]=XY[threadIdx.x+blockDim.x]; 
}

void ch8_ex_scan(real* d_X, real* d_Y,int inputsize){
	ch8_ex_scan_kernel<<<1,SECTION_SIZE/2>>>(d_X,d_Y,inputsize);
	gpuErrchk(hipPeekAtLastError());
}
