#include "gpuerrchk.cuh"
#include "assert.h"
#include "real.h"
#include "ch8_ex_scan.cuh"
#include <iostream>
#define DAT_SIZE 8
void test(){
	real X[DAT_SIZE];
	real Y[DAT_SIZE];
	real* d_X;
	real* d_Y;
	gpuErrchk(hipMalloc((void**) &d_X,sizeof(real)*DAT_SIZE));
	gpuErrchk(hipMemcpy(d_X,X,sizeof(real)*DAT_SIZE,hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**) &d_Y,sizeof(real)*DAT_SIZE ));
	ch8_ex_scan(d_X,d_Y,8);
	gpuErrchk( hipPeekAtLastError() );		
	gpuErrchk(hipMemcpy(Y,d_Y,sizeof(real)*DAT_SIZE,hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_X));
	gpuErrchk(hipFree(d_Y));

}

int main(){
	test();
}

