#include "hip/hip_runtime.h"
#include "ppm.h"
const int BLUR_SIZE=10;
//used to blur a 2d color ppm image
__global__	
void blurKernel(int* in, int* out,int w,int h){
	int curpix=blockIdx.x*blockDim.x+threadIdx.x;
	int row=curpix / w;
	int col=curpix % w;
	if ( row>=h ) return;
	int pixr=0;
	int pixg=0;
	int pixb=0;
	int pixels=0;
	for (int br=-BLUR_SIZE; br<=BLUR_SIZE;br++){
	for (int bc=-BLUR_SIZE; bc<=BLUR_SIZE;bc++){
		int currow=row+br;
		int curcol=col+bc;
		if(currow>=0 && currow<h && curcol>=0 && curcol <w ){
			pixels++;
			int pdex= 3*(currow*w+curcol);
			pixr+=in[pdex];
			pixg+=in[pdex+1];
			pixb+=in[pdex+2];
		}
	}
	}
	int dex=3*(row*w+col);
	out[dex]= round((float)pixr/pixels);
	out[dex+1]= round((float)pixg/pixels);
	out[dex+2]= round((float)pixb/pixels);
}

int main(){
	ppm football("football.ppm");
	int numpixels=football.height*football.width;
	int size=3*numpixels;
	int arsize=sizeof(int)*size;
	std::cout <<"Size is: "<< size;
	int* d_football_data;
	int* d_bfootball_data;
	hipMalloc((void**)&d_football_data,arsize);
	hipMalloc((void**)&d_bfootball_data,arsize );
	hipMemcpy(d_football_data,football.data,arsize,hipMemcpyHostToDevice);
	hipMemcpy(d_bfootball_data,football.data,arsize,hipMemcpyHostToDevice);

	blurKernel<<<ceil(numpixels/256) ,256>>>(d_football_data,d_bfootball_data,football.width,football.height);

	ppm bfootball(football);
	hipMemcpy(bfootball.data,d_bfootball_data,arsize,hipMemcpyDeviceToHost);
	bfootball.write("bfootball.ppm");
	hipFree(d_football_data);
	hipFree(d_bfootball_data);
}
