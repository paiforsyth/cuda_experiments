#include "hip/hip_runtime.h"
#include "ppm.h"
#include <math.h>
#include <iostream>
#include <stdio.h>
__global__
void colorToGreyScaleConversion(int* imdata,int* outimdata,int size){
	int dex= 3*(threadIdx.x+blockIdx.x*blockDim.x);

	if (dex>= size) return;
	int r=imdata[dex];
	int g=imdata[dex+1];
	int b=imdata[dex+2];
	int grey= round(255*( 0.21*(r/255.0)+0.71*(g/255.0)+0.07*(b/255.0)));
//	printf("Grey value is : ")
	outimdata[dex]=grey;
	outimdata[dex+1]=grey;
	outimdata[dex+2]=grey;
	
}
int main(){
	ppm football("football.ppm");
	int size=3*football.height*football.width;
	int arsize=sizeof(int)*size;
	std::cout <<"Size is: "<< size;
	int* d_football_data;
	int* d_gfootball_data;
	hipMalloc((void**)&d_football_data,arsize);
	hipMalloc((void**)&d_gfootball_data,arsize );
	hipMemcpy(d_football_data,football.data,arsize,hipMemcpyHostToDevice);
	hipMemcpy(d_gfootball_data,football.data,arsize,hipMemcpyHostToDevice);

	colorToGreyScaleConversion<<< 1<<20 ,256>>>(d_football_data,d_gfootball_data,size);
	ppm gfootball(football);
	hipMemcpy(gfootball.data,d_gfootball_data,arsize,hipMemcpyDeviceToHost);
	gfootball.write("gfootball.ppm");
	hipFree(d_gfootball_data);
	hipFree(d_football_data);
}
