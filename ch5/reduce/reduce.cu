#include "hip/hip_runtime.h"
#include "real.h"
#include "gpuerrchk.cuh"
#include "reduce.cuh"
//kernel used to sum an array containing size elements, where size can be 2^0,2^1,..,2^10
//intended to be called as a single thread block with size threads

//Note: here is the reason that the below operation never reads outside the bounds of partialsum if it has length 2^k.
//since t%(2*stride)==0 we have that  t=m*2^{i+1} for some m and where we are in iteration i (stride=2^i).  Since t is in the bounds of the array,
//t<2^k. 
//Thus 2m<2^{k-i} (*).
//But then t+2^i<2^{k} if and only if (2m+1)<2^{k-i}.  This holds since both sides of equation * are even.
__global__
void kreducev1(real* X, real* d_ans, unsigned int size){
	extern __shared__ real partialsum[];	
	unsigned int t= threadIdx.x;
	partialsum[t]=X[t];
	for (unsigned int  stride = 1; stride < size; stride *= 2 ){
		__syncthreads();
		if (t % (2*stride) == 0)
			partialsum[t]+=partialsum[t+stride];
	}
	if (t == 0)
		*d_ans=partialsum[0];
}

__global__
void kreducev2(real* X, real* d_ans, unsigned int size){
	extern __shared__ real partialsum[];	
	unsigned int t= threadIdx.x;
	partialsum[t]=X[t];
	for (unsigned int stride = size/2; stride >= 1; stride = stride >> 1){
		__syncthreads();
		if (t < stride)
			partialsum[t] += partialsum[t+stride];
	}
	
	if (t == 0)
		*d_ans=partialsum[0];

}

real reducev1(real* X, unsigned int numel){
	int memsize= sizeof(real)*numel;
	real* d_X;
	real* d_ans;
	gpuErrchk(hipMalloc( (void**) &d_X, memsize));
	gpuErrchk(hipMalloc( (void**) &d_ans, sizeof(real)  ) );
	gpuErrchk(hipMemcpy(d_X, X, memsize, hipMemcpyHostToDevice));
	kreducev1<<<1, numel, memsize>>>(d_X,d_ans,numel);
	gpuErrchk( hipPeekAtLastError() );	
	gpuErrchk( hipFree(d_X));
	real ans;
	gpuErrchk(hipMemcpy(&ans,d_ans,sizeof(real),hipMemcpyDeviceToHost ));
	gpuErrchk(hipFree(d_ans));
	return ans;
}



real reducev2(real* X, unsigned int numel){
	int memsize= sizeof(real)*numel;
	real* d_X;
	real* d_ans;
	gpuErrchk(hipMalloc( (void**) &d_X, memsize));
	gpuErrchk(hipMalloc( (void**) &d_ans, sizeof(real)  ) );
	gpuErrchk(hipMemcpy(d_X, X, memsize, hipMemcpyHostToDevice));
	kreducev2<<<1, numel, memsize>>>(d_X,d_ans,numel);
	gpuErrchk( hipPeekAtLastError() );	
	gpuErrchk( hipFree(d_X));
	real ans;
	gpuErrchk(hipMemcpy(&ans,d_ans,sizeof(real),hipMemcpyDeviceToHost ));
	gpuErrchk(hipFree(d_ans));
	return ans;
}
