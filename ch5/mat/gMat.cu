#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <assert.h>
#include <memory>
#include <random>
#include "gMat.cuh"
#include "gpuerrchk.cuh"
#include "real.h"
__global__ void matMulKernel(real* A, real* B, real* P, int m, int  n, int s, int tile_size){
	//each thread in the block will be responsible for a different element of these arrays
	extern __shared__ real tilemem[];
	real* Atile=tilemem;
	real* Btile=&tilemem[tile_size*tile_size];

	int b1=blockIdx.y;
	int b2=blockIdx.x;
	int t1=threadIdx.y;
	int t2=threadIdx.x;

	int row=b1*tile_size+t1;// the indices of the element of P this thread is responsible for computing
	int col=b2*tile_size+t2;

	real Pvalue=0;
	//loop over the tiles necessary to compute the element of P of interest
	for (int j=0; j<ceil( n / (real) tile_size);j++){
		//collaborate with other threads to store the current tiles of A and B
		//this thread is responsible for storing  the (t_1,t_2) element of the (b_1,j) tile of A and the (j,b_2) tile of B
		if( (b1*tile_size+t1) < m && (j*tile_size+t2)<n )
			Atile[tile_size*t1 + t2]=A[ n*(b1*tile_size+t1) + j*tile_size  + t2 ]; // We access A[b1*tile_size+t1][j*tile_size+t2] using linearized indices (A has n columns)
		else
			Atile[tile_size*t1 + t2]=0;
		if( (j*tile_size+t1)<n && (b2*tile_size+t2)<s  )
			Btile[tile_size*t1 + t2]=B[ s*(j*tile_size+t1)  + b2*tile_size + t2 ]; //we access the B[j*tile_size+t1][b2*tile_size+t2] element using linearized indices (B has s columns)
		else
			Btile[tile_size*t1 + t2]=0;
		__syncthreads();

		//sum the elements of the current A and B tiles used in the computation of the element of P for which this thread is responsible
		for (int k=0; k<tile_size; k++)
			Pvalue += Atile[tile_size*t1 + k]*Btile[tile_size*k + t2];

		__syncthreads();

	}
	if(row<m && col<s) P[row*s + col]=Pvalue;

}
//stretched version
//stretches the tiles by a factor of stretch in one direction, so that each thread computes stretch entries of P
//thus more memory need to be allocated when invoking this kernel
__global__ void matMulKernelv2(real* A, real* B, real* P, int m, int  n, int s, int tile_size, int stretch){
	//each thread in the block will be responsible for  different elements of these arrays

	extern __shared__ real tilemem[];
	real* Atile=tilemem;// 2d dimensions of Atile should be thought of as tile_size by stretch*tile_size
	real* Btile=&tilemem[stretch*tile_size*tile_size]; 

	int b1=blockIdx.y;
	int b2=blockIdx.x;
	int t1=threadIdx.y;
	int t2=threadIdx.x;

	int row=b1*tile_size+t1;
	int col=b2*tile_size+t2;

	real Pvalue=0;
	//loop over the tiles necessary to compute the element of P of interest
	for (int j=0; j<ceil( n / (real) tile_size / stretch  );j++){
		//collaborate with other threads to store the current tiles of A and B
		for (int q=0; q != stretch; ++q){

			if( (b1*tile_size+t1) < m && (j*stretch*tile_size + q*tile_size + t2) < n )
				Atile[stretch*tile_size*t1 + q*tile_size + t2]=A[ n*(b1*tile_size+t1) + j*stretch*tile_size + q*tile_size   + t2 ]; // We access A[b1*tile_size+t1][j*stretch*tile_size+q*tile_size+t2] using linearized indices (A has n columns)
			else
				Atile[stretch*tile_size*t1 + q*tile_size + t2]=0;
			if( (j*stretch*tile_size + q*tile_size  +t1)<n && (b2*tile_size+t2)<s  )
				Btile[tile_size*t1 + q*tile_size*tile_size + t2]=B[ s*(j*stretch*tile_size+q*tile_size+t1)  + b2*tile_size + t2 ]; //we access the B[j*stretch*tile_size+q*tile_size+t1][b2*tile_size+t2] element using linearized indices (B has s columns)
			else
				Btile[tile_size*t1 + q*tile_size*tile_size + t2]=0;

		}

		__syncthreads();

		//sum the elements of the current A and B tiles used in the computation of the element of P for which this thread is responsible
		for (int k=0; k<stretch*tile_size; k++)
			Pvalue += Atile[stretch*tile_size*t1 + k]*Btile[tile_size*k + t2];

		__syncthreads();

	}
	if(row<m && col<s) P[row*s + col]=Pvalue;

}



gMat::gMat(std::vector<real> datavector, int r, int c,int devnumber): rows{r}, cols{c} {
	assert(datavector.size() == rows*cols );
	int size=sizeof(real)*rows*cols;

	hipDeviceProp_t dev_prop;	
	hipGetDeviceProperties(&dev_prop,devnumber);
	assert(dev_prop.totalGlobalMem>size);

	gpuErrchk(hipMalloc( (void**)&d_data, size));
	gpuErrchk(hipMemcpy(d_data, datavector.data(), size, hipMemcpyHostToDevice));
}

void gMat::cleanup(){
	gpuErrchk(hipFree(d_data));
}



int gMat::getrows(){ return rows;}
int gMat::getcols(){ return cols;}

real gMat::entry(int i, int j){
	assert(i>=0 && i<rows);
	assert(j>=0 && j<cols);
	real val=0;

	hipMemcpy(&val, &d_data[i*cols+j],sizeof(real),hipMemcpyDeviceToHost );
	return val;
}


std::vector<real> gMat::tovector(){
	real* h_data=new real[rows*cols];
	int size=rows*cols*sizeof(real);
	hipMemcpy(h_data,d_data,size,hipMemcpyDeviceToHost);
	std::vector<real> v;
	for (int i=0; i<rows; ++i) for (int j=0; j<cols; ++j){
		v.push_back(h_data[i*cols+j]);
	}
	delete[] h_data;
	return v;
}	

gMat randgMat(int r, int c){
	std::vector<real> datavector;
	std::default_random_engine re;
	std::uniform_real_distribution<real> dist{0.0,1.0};
	for (int i=0; i<r*c; i++) datavector.push_back(dist(re));
	return gMat(datavector,r,c);
}

gMat eye(int r){
	std::vector<real> datavector;
	for (int i=0; i!=r; ++i) for(int j=0; j!=r; ++j) 
		if(i==j) 
			datavector.push_back(1.0);
		else
			datavector.push_back(0.0);
	return gMat(datavector,r,r);
}

void prod(const gMat& A, const gMat& B, gMat& P) {
	int w=16;
	int mem=2*w*w*sizeof(real);
	prod(A,B,P,mem);
}

void prod(const gMat& A, const gMat& B, gMat& P, int mem){
	assert(A.cols==B.rows && A.rows==P.rows && B.cols==P.cols);
	int tile_size= floor(sqrt( mem/2/sizeof(real)));
	assert(tile_size>0);	

	matMulKernel<<<dim3(ceil(B.cols/ (real) tile_size), ceil( A.rows/ (real) tile_size)), dim3(tile_size,tile_size), mem>>>(A.d_data, B.d_data, P.d_data,A.rows, A.cols, B.cols, tile_size );
	gpuErrchk( hipPeekAtLastError() );	
}

void prodv2(const gMat& A, const gMat& B, gMat& P, int tile_size, int stretch){
	assert(A.cols==B.rows && A.rows==P.rows && B.cols==P.cols);
	assert(tile_size>0 && stretch>0);
	int mem= 2*tile_size*tile_size*stretch*sizeof(real);
	matMulKernelv2 <<<dim3(ceil(B.cols/ (real) tile_size), ceil( A.rows/ (real) tile_size)), dim3(tile_size,tile_size), mem>>>(A.d_data, B.d_data, P.d_data, A.rows, A.cols, B.cols, tile_size, stretch);
	gpuErrchk( hipPeekAtLastError() );	
}

std::ostream& operator<<(std::ostream& os,  gMat& gm)  
{  
	std::vector<real> v=gm.tovector();
	for (int i=0; i<gm.rows; ++i){
		for (int j=0; j<gm.cols; ++j){
			os << v[i*gm.cols+j] << ",";
		}
		os << "\n";
	}
	return os;  

}  


