#include "hip/hip_runtime.h"
#include <iostream>
#include "gMat.cuh"
#include <vector>
#include "mytime.h"
#include <functional>
#include <math.h>
#include <stdlib.h> 
#include <string>
#include "real.h"
void manyMult(gMat& A, gMat& B, gMat& C,int iter,int mem){
	for (int i=0; i!=iter; ++i){
		prod(A,B,C,mem);
		prod(A,C,B,mem);
	}
}

void manyMultv2(gMat& A, gMat& B, gMat& C,int iter,int tile_size, int stretch ){
	for (int i=0; i!=iter; ++i){
		prodv2(A,B,C,tile_size,stretch);
		prodv2(A,C,B,tile_size,stretch);
	}
}

void multstat(int mem, int row, int col){
	std::cout << "multiplying "  << row << " by "<< col <<" matrices.\n";
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	std::cout << "using "<< mem << " of "<<prop.sharedMemPerBlock << "shared memory\n";
	int w=floor(sqrt(mem/2/sizeof(real)));
	std::cout << "Tile sizes will be: "<< w<< " by "<< w << "\n";
	std::cout << "This amounts to: " << w*w << " threads \n";
	std::cout <<  ceil(row/ (real) w)*ceil(col/ (real) w) << " tiles will be used "<<std::endl;
}

int main(int argc, char *argv[]){
	int w;
	if ( argc >= 2) w=atoi(argv[1]);
	else w=20;
	int r=1000;
	int c=1000;
	gMat A=eye(r);
	gMat B=randgMat(r,c);
	gMat C=randgMat(r,c);
//	std::cout<< "(5,5) entry of A is: "<< A.entry(5,5)<< std::endl;
	A.name="A";
	B.name="B";
	C.name="C";
	//int mem=2*w*w*sizeof(real);
	auto mm=std::bind(manyMultv2,A,B,C,300,w,4);
	//multstat(mem,r,c);
	mm();
	A.cleanup();
	B.cleanup();
	C.cleanup();
	 
}




void test1(){
	std::vector<real> adat={1,2,3,4};
	std::vector<real> bdat={2,0,0,2,0,2};
	std::vector<real> cdat={0,0,0,0,0,0};
	gMat A{adat,2,2};
	gMat B{bdat,2,3};
	gMat C{cdat,2,3};
	std::cout <<"A:\n" << A << std::flush;
	std::cout <<"B:\n" << B << std::flush;
	int mem=8*sizeof(real);
	prod(A,B,C,mem);
	std::cout <<"C:\n" << C << std::flush;
}
